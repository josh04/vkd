#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    if (y < height / 2) {
        pos[y * width + x] = make_float4(u, w, v, 1.0f);
    }
}

void run_simple_vbo_kernel(dim3 block, dim3 grid, float4* pos, unsigned int width, unsigned int height, float time) {
    simple_vbo_kernel<<<grid, block>>>(pos, width, height, time);
}

__global__ void simple_image_kernel(hipTextureObject_t colour, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    float4 c = make_float4(u, w, v, 1.0f);
    surf2Dwrite(c, colour, x * sizeof(float4), y);
}

#include "GL/gl3w.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"

hipTextureObject_t inTexObject;

void run_simple_image_kernel(dim3 block, dim3 grid, hipArray_t colour, unsigned int width, unsigned int height, float time) {

    struct hipChannelFormatDesc desc;
    checkCudaErrors(hipGetChannelDesc(&desc, colour));

    hipResourceDesc            texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = colour;

    hipTextureDesc             texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&inTexObject, &texRes, &texDescr, NULL));

    simple_image_kernel<<<grid, block>>>(inTexObject, width, height, time);
}

__global__ void buffer_to_image_kernel(float4* col, hipTextureObject_t colour, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    /*
    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    float4 c = make_float4(u, w, v, 1.0f);
*/

    surf2Dwrite(col[y * width + x], colour, x * sizeof(float4), y);
}
void run_buffer_to_image_kernel(dim3 block, dim3 grid, float4* col, hipArray_t colour, unsigned int width, unsigned int height, float time) {

    struct hipChannelFormatDesc desc;
    checkCudaErrors(hipGetChannelDesc(&desc, colour));

    hipResourceDesc            texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = colour;

    hipTextureDesc             texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&inTexObject, &texRes, &texDescr, NULL));

    buffer_to_image_kernel<<<grid, block>>>(col, inTexObject, width, height, time);
}